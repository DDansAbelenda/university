
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <chrono>

#define ThreadsPerBlock 256

// Función a integrar
__host__ __device__ double function(double x)
{
    return x * x * x; // f(x) = x^3
}

// Función para realizar una suma atómica de tipo double en el dispositivo (GPU)
__device__ double atomicAddDouble(double *address, double val)
{
    // Convierte el puntero a dirección de memoria de tipo double a un puntero de tipo unsigned long long int
    unsigned long long int *address_as_ull = (unsigned long long int *)address;

    // Lee el valor actual de la dirección de memoria y guarda el valor leído en 'old'
    unsigned long long int old = *address_as_ull, assumed;

    // Bucle do-while para realizar la suma atómica de manera segura
    do
    {
        // Asigna el valor actual de 'old' a 'assumed'
        assumed = old;

        // Utiliza atomicCAS para realizar una suma atómica en punto flotante de doble precisión
        // __double_as_longlong convierte un double a un long long int, realiza la suma y luego convierte el resultado de vuelta a double
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));

        // Continúa el bucle si el valor asumido ('assumed') no coincide con el valor actual ('old')
    } while (assumed != old);

    // Convierte el valor final en 'old' de tipo unsigned long long int a double y lo devuelve
    return __longlong_as_double(old);
}

// Kernel para el cálculo de la integral usando el método del trapecio de forma paralela
__global__ void trapecioParallel(double a, double b, double h, int n, double *d_result)
{
    // Calcula el índice global del hilo
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Verifica si el índice supera el número total de intervalos
    if (idx >= n)
        return;

    // Índice del hilo dentro del bloque
    const int s_idx = threadIdx.x;

    // Memoria compartida para almacenar temporalmente resultados de la función para el bloque
    __shared__ double buffer_per_block[ThreadsPerBlock];

    // Calcula el valor de x para el punto actual del intervalo
    double x = a + idx * h;

    // Evalúa la función en el punto x y almacena el resultado en memoria compartida
    buffer_per_block[s_idx] = function(x);

    __syncthreads();

    if (s_idx == 0)
    {
        double blockSum = 0;
        // Suma los resultados del bloque
        for (int i = 0; i < blockDim.x; ++i)
        {
            blockSum += buffer_per_block[i];
        }

        // Realiza una suma atómica en el resultado global
        atomicAddDouble(d_result, blockSum);
    }
}

// Función que se encarga de crear las condiciones para lanzar el kernel y lo activa
void launcherKernel(double a, double b, double h, int n)
{
    // Reservar memoria en la GPU para el resultado
    double *d_result;
    hipMalloc((void **)&d_result, sizeof(double));
    // Inicializa el resultado en la GPU con el valor correspondiente al método del trapecio para los extremos del intervalo
    double initResult = 0.5 * (function(a) + function(b));
    hipMemset(d_result, initResult, sizeof(double));

    // Configuración de bloques e hilos en la GPU
    int blockSize = ThreadsPerBlock;                // hilos por bloque
    int gridSize = (n + blockSize - 1) / blockSize; // bloques por grid

    // Lanzar el kernel en la GPU
    trapecioParallel<<<gridSize, blockSize>>>(a, b, h, n, d_result);

    // Copiar el resultado desde la GPU a la CPU
    double result;
    hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    // Imprimir el resultado
    printf("Resultado de la integral: %f \n", result * h);
    // Liberar memoria en la GPU
    hipFree(d_result);
}

// Función para el cálculo de la integral usando el método del trapecio de forma secuencial
void trapecio(double a, double b, double h, int n)
{

    double result = 0.5 * (function(a) + function(b));

    for (int i = 1; i < n; i++)
    {

        double x = a + i * h;

        result += function(x);
    }
}

int main(int argc, char *argv[])
{
    // Verificar los parámetros
    if (argc != 4)
    {
        std::cerr << "Uso: " << argv[0] << " <a> <b> <n_intervalos>" << std::endl;
        return 1;
    }

    // Inicialización de los intervalos y la cantidad de trapecios
    double a = atof(argv[1]);
    double b = atof(argv[2]);
    int n = atoi(argv[3]);
    double h = (b - a) / n; // calcula el ancho del intervalo

    // Trabajo en Paralelo
    printf("Calculo en Paralelo\n");

    //  Crear eventos para medir el tiempo
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Registrar el evento de inicio
    hipEventRecord(start, 0);

    // Lanzar el cálculo paralelo
    launcherKernel(a, b, h, n);

    // Registrar el evento de parada
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calcular y imprimir el tiempo de ejecución
    float parallelTime = 0;
    hipEventElapsedTime(&parallelTime, start, stop);
    printf("Tiempo de ejecución: %f ms\n", parallelTime);

    // Dividir
    printf("--------------------------\n");

    // Trabajo secuencial
    printf("Calculo Secuencial\n");

    // Medir el tiempo de ejecución
    clock_t init_time = clock();

    // Llamar a la función
    trapecio(a, b, h, n);

    // Calcular e imprimir el tiempo transcurrido en milisegundos
    clock_t finish_time = clock();
    float sequentialTime = (float)(finish_time - init_time) / CLOCKS_PER_SEC;

    printf("Tiempo de ejecución: %f ms\n", sequentialTime * 1000);

    printf("----------------------\n");

    printf("Para N = %d el speedup es: %f \n", n, (sequentialTime * 1000) / parallelTime);

    return 0;
}
